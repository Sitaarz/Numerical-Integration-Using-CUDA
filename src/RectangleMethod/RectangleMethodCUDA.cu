//
// Created by Krystian on 10.05.2025.
//

#include <stdexcept>
#include <hip/hip_runtime.h>
#include <iostream>

#include "RectangleMethodCUDA.h"
#include "RectangleKernel.cuh"
#include "../Constants.cuh"
#include "../../common/Kernels.cuh"

double RectangleMethodCUDA::calculate(FunctionType functionType, double a, double b, int n, bool test) {
    if (n <= 0) throw std::invalid_argument("n must be positive");
    if (b <= a) throw std::invalid_argument("b must be greater than a");

    hipEventRecord(start, 0);

    double* d_results;
    double delta = (b - a) / n;
    hipError_t error = hipMalloc(&d_results, n * sizeof(double));
    if (error != hipSuccess) throw std::runtime_error("Failed to allocate device memory: " + std::string(hipGetErrorString(error)));

    int blocksPerGrid = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
    rectangleKernel<<<blocksPerGrid, BLOCK_SIZE>>>(functionType, delta, a, n, d_results);
    error = hipGetLastError();
    if (error != hipSuccess) { hipFree(d_results); throw std::runtime_error("Failed to launch kernel: " + std::string(hipGetErrorString(error))); }
    error = hipDeviceSynchronize();
    if (error != hipSuccess) { hipFree(d_results); throw std::runtime_error("Failed to synchronize device: " + std::string(hipGetErrorString(error))); }

    // Redukcja sumy na GPU
    double* d_sum;
    hipMalloc(&d_sum, sizeof(double));
    int reduce_n = n;
    double* d_in = d_results;
    double* d_out = nullptr;
    while (reduce_n > 1) {
        int reduce_blocks = (reduce_n + BLOCK_SIZE - 1) / BLOCK_SIZE;
        d_out = (reduce_blocks == 1) ? d_sum : d_results; // ostatnia iteracja do d_sum
        reduceSumKernel<<<reduce_blocks, BLOCK_SIZE>>>(d_in, d_out, reduce_n);
        hipDeviceSynchronize();
        d_in = d_out;
        reduce_n = reduce_blocks;
    }

    double integral = 0.0;
    hipMemcpy(&integral, d_sum, sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_results);
    hipFree(d_sum);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&timeElapsed, start, stop);

    if (test) {
        std::cout << "Time " << timeElapsed << " ms" << std::endl;
    }

    return integral;
}