//
// Created by Krystian on 10.05.2025.
//

#include <stdexcept>
#include <hip/hip_runtime.h>
#include "RectangleMethodCUDA.h"
#include "RectangleKernel.cuh"

double RectangleMethodCUDA::calculate(FunctionType functionType, double a, double b, int n) {
    if (n <= 0) {
        throw std::invalid_argument("n must be positive");
    }
    if (b <= a) {
        throw std::invalid_argument("b must be greater than a");
    }

    double* d_results;
    double delta = (b - a) / n;

        hipError_t error = hipMalloc(&d_results, n * sizeof(double));
        if (error != hipSuccess) {
            throw std::runtime_error("Failed to allocate device memory: " + std::string(hipGetErrorString(error)));
        }

        int threadsPerBlock = 256;
        int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

        rectangleKernel<<<blocksPerGrid, threadsPerBlock>>>(functionType, delta, a, n, d_results);
        error = hipGetLastError();
        if (error != hipSuccess) {
            hipFree(d_results);
            throw std::runtime_error("Failed to launch kernel: " + std::string(hipGetErrorString(error)));
        }
        
        error = hipDeviceSynchronize();
        if (error != hipSuccess) {
            hipFree(d_results);
            throw std::runtime_error("Failed to synchronize device: " + std::string(hipGetErrorString(error)));
        }

        auto* h_results = new double[n];
        error = hipMemcpy(h_results, d_results, n * sizeof(double), hipMemcpyDeviceToHost);
        if (error != hipSuccess) {
            delete[] h_results;
            hipFree(d_results);
            throw std::runtime_error("Failed to copy results from device: " + std::string(hipGetErrorString(error)));
        }

        double integral = 0;
        for (int i = 0; i < n; i++) {
            integral += h_results[i];
        }

        delete[] h_results;
        hipFree(d_results);

    return integral;
}