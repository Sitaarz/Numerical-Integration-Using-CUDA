#include <stdexcept>
#include <hip/hip_runtime.h>
#include "MonteCarloMethodCUDA.cuh"
#include "MonteCarloKernel.cuh"
#include "../Constants.cuh"

double MonteCarloMethodCUDA::calculate(FunctionType functionType, double a, double b, int n, bool test) {
    if (n <= 0) {
        throw std::invalid_argument("n must be positive");
    }
    if (b <= a) {
        throw std::invalid_argument("b must be greater than a");
    }

    double* d_results;
    hipError_t error = hipMalloc(&d_results, n * sizeof(double));
    if (error != hipSuccess) {
        throw std::runtime_error("Failed to allocate device memory: " + std::string(hipGetErrorString(error)));
    }

    int blocksPerGrid = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    monteCarloKernel<<<blocksPerGrid, BLOCK_SIZE>>>(functionType, a, b, n, d_results);

    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float ms = 0;
    hipEventElapsedTime(&ms , start , stop);

    if (test) std::cout << "Time: " << ms << " ms" << std::endl;

    error = hipGetLastError();
    if (error != hipSuccess) {
        hipFree(d_results);
        throw std::runtime_error("Failed to launch kernel: " + std::string(hipGetErrorString(error)));
    }

    error = hipDeviceSynchronize();
    if (error != hipSuccess) {
        hipFree(d_results);
        throw std::runtime_error("Failed to synchronize device: " + std::string(hipGetErrorString(error)));
    }

    auto* h_results = new double[n];
    error = hipMemcpy(h_results, d_results, n * sizeof(double), hipMemcpyDeviceToHost);
    if (error != hipSuccess) {
        delete[] h_results;
        hipFree(d_results);
        throw std::runtime_error("Failed to copy results from device: " + std::string(hipGetErrorString(error)));
    }

    double sum = 0.0;
    for (int i = 0; i < n; i++) {
        sum += h_results[i];
    }

    double integral = (b - a) * sum / n;

    delete[] h_results;
    hipFree(d_results);

    return integral;
}